#include "hip/hip_runtime.h"
  __global__ void computeTriangleStuff(float *dPoints, float* dResults, int num_points, float x, float y, float z, float qx, float qy, float qz, float qw) {
    int idx = ThreadIdx.x + blockIdx.x * blockDim.x;
    
    if(idx * 12 < num_points) {
        dResults[idx * 4] = 0;
        dResults[idx * 4 + 1] = 0;//etc.
        
        //All the triangle computation goes
        
        
      }
  }